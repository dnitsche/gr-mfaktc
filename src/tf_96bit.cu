#include "hip/hip_runtime.h"
/*
This file is part of mfaktc.
Copyright (C) 2009, 2010, 2011, 2012  Oliver Weihe (o.weihe@t-online.de)

mfaktc is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

mfaktc is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.
                                
You should have received a copy of the GNU General Public License
along with mfaktc.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>  

#include "params.h"
#include "my_types.h"
#include "compatibility.h"
#include "my_intrinsics.h"

#define NVCC_EXTERN
#include "sieve.h"
#include "timer.h"
#include "output.h"
#undef NVCC_EXTERN

#include "tf_debug.h"
#include "tf_96bit_base_math.cu"
#include "tf_96bit_helper.cu"


#ifndef CHECKS_MODBASECASE
__device__ static void mod_192_96(int96 *res, int192 q, int96 n, float nf)
#else
__device__ static void mod_192_96(int96 *res, int192 q, int96 n, float nf, unsigned int *modbasecase_debug)
#endif
/* res = q mod n */
{
  float qf;
  unsigned int qi;
  int192 nn;

/********** Step 1, Offset 2^75 (2*32 + 11) **********/
/*
the 75 bit kernel has only one difference: the first iteration of the
division will be skipped
*/
#ifndef SHORTCUT_75BIT
  qf= __uint2float_rn(q.d5);
  qf= qf * 4294967296.0f + __uint2float_rn(q.d4);
  qf= qf * 4294967296.0f + __uint2float_rn(q.d3);
  qf*= 2097152.0f;

  qi=__float2uint_rz(qf*nf);

  MODBASECASE_QI_ERROR(1<<22, 1, qi, 0);


// nn = n * qi
  nn.d2 =                                 __umul32(n.d0, qi);
  nn.d3 = __add_cc (__umul32hi(n.d0, qi), __umul32(n.d1, qi));
  nn.d4 = __addc_cc(__umul32hi(n.d1, qi), __umul32(n.d2, qi));
  nn.d5 = __addc   (__umul32hi(n.d2, qi),                  0);

// shiftleft nn 11 bits
  nn.d5 = (nn.d5 << 11) + (nn.d4 >> 21);
  nn.d4 = (nn.d4 << 11) + (nn.d3 >> 21);
  nn.d3 = (nn.d3 << 11) + (nn.d2 >> 21);
  nn.d2 =  nn.d2 << 11;

//  q = q - nn
  q.d2 = __sub_cc (q.d2, nn.d2);
  q.d3 = __subc_cc(q.d3, nn.d3);
  q.d4 = __subc_cc(q.d4, nn.d4);
  q.d5 = __subc   (q.d5, nn.d5);
#endif // SHORTCUT_75BIT
/********** Step 2, Offset 2^55 (1*32 + 23) **********/
#ifndef SHORTCUT_75BIT  
  qf= __uint2float_rn(q.d5);
  qf= qf * 4294967296.0f + __uint2float_rn(q.d4);
#else
  qf= __uint2float_rn(q.d4);
#endif  
  qf= qf * 4294967296.0f + __uint2float_rn(q.d3);
  qf= qf * 4294967296.0f + __uint2float_rn(q.d2);
  qf*= 512.0f;

  qi=__float2uint_rz(qf*nf);

  MODBASECASE_QI_ERROR(1<<22, 2, qi, 1);


// nn = n * qi
  nn.d1 =                                 __umul32(n.d0, qi);
  nn.d2 = __add_cc (__umul32hi(n.d0, qi), __umul32(n.d1, qi));
  nn.d3 = __addc_cc(__umul32hi(n.d1, qi), __umul32(n.d2, qi));
  nn.d4 = __addc   (__umul32hi(n.d2, qi),                  0);

// shiftleft nn 23 bits
#ifdef CHECKS_MODBASECASE
  nn.d5 =                  nn.d4 >> 9;
#endif  
  nn.d4 = (nn.d4 << 23) + (nn.d3 >> 9);
  nn.d3 = (nn.d3 << 23) + (nn.d2 >> 9);
  nn.d2 = (nn.d2 << 23) + (nn.d1 >> 9);
  nn.d1 =  nn.d1 << 23;

// q = q - nn
  q.d1 = __sub_cc (q.d1, nn.d1);
  q.d2 = __subc_cc(q.d2, nn.d2);
  q.d3 = __subc_cc(q.d3, nn.d3);
#ifndef CHECKS_MODBASECASE  
  q.d4 = __subc   (q.d4, nn.d4);
#else
  q.d4 = __subc_cc(q.d4, nn.d4);
  q.d5 = __subc   (q.d5, nn.d5);
#endif

/********** Step 3, Offset 2^35 (1*32 + 3) **********/
  MODBASECASE_NONZERO_ERROR(q.d5, 3, 5, 2);

  qf= __uint2float_rn(q.d4);
  qf= qf * 4294967296.0f + __uint2float_rn(q.d3);
  qf= qf * 4294967296.0f + __uint2float_rn(q.d2);
  qf*= 536870912.0f; // add (q.d1 >> 3) ???
//  qf*= 4294967296.0f; /* this includes the shiftleft of qi by 3 bits! */

  qi=__float2uint_rz(qf*nf);

  MODBASECASE_QI_ERROR(1<<22, 3, qi, 3);

// shiftleft qi 3 bits to avoid "long shiftleft" after multiplication
  qi <<= 3;

// nn = n * qi
  nn.d1 =                                 __umul32(n.d0, qi);
  nn.d2 = __add_cc (__umul32hi(n.d0, qi), __umul32(n.d1, qi));
  nn.d3 = __addc_cc(__umul32hi(n.d1, qi), __umul32(n.d2, qi));
  nn.d4 = __addc   (__umul32hi(n.d2, qi),                  0);

//  q = q - nn
  q.d1 = __sub_cc (q.d1, nn.d1);
  q.d2 = __subc_cc(q.d2, nn.d2);
  q.d3 = __subc_cc(q.d3, nn.d3);
  q.d4 = __subc   (q.d4, nn.d4);

/********** Step 4, Offset 2^15 (0*32 + 15) **********/
  MODBASECASE_NONZERO_ERROR(q.d5, 4, 5, 4);

  qf= __uint2float_rn(q.d4);
  qf= qf * 4294967296.0f + __uint2float_rn(q.d3);
  qf= qf * 4294967296.0f + __uint2float_rn(q.d2);
  qf= qf * 4294967296.0f + __uint2float_rn(q.d1);
  qf*= 131072.0f;
  
  qi=__float2uint_rz(qf*nf);

  MODBASECASE_QI_ERROR(1<<22, 4, qi, 5);

// nn = n * qi
  nn.d0 =                                 __umul32(n.d0, qi);
  nn.d1 = __add_cc (__umul32hi(n.d0, qi), __umul32(n.d1, qi));
  nn.d2 = __addc_cc(__umul32hi(n.d1, qi), __umul32(n.d2, qi));
  nn.d3 = __addc   (__umul32hi(n.d2, qi),                  0);

// shiftleft nn 15 bits
#ifdef CHECKS_MODBASECASE
  nn.d4 =                  nn.d3 >> 17;
#endif
  nn.d3 = (nn.d3 << 15) + (nn.d2 >> 17);
  nn.d2 = (nn.d2 << 15) + (nn.d1 >> 17);
  nn.d1 = (nn.d1 << 15) + (nn.d0 >> 17);
  nn.d0 =  nn.d0 << 15;

//  q = q - nn
  q.d0 = __sub_cc (q.d0, nn.d0);
  q.d1 = __subc_cc(q.d1, nn.d1);
  q.d2 = __subc_cc(q.d2, nn.d2);
#ifndef CHECKS_MODBASECASE
  q.d3 = __subc   (q.d3, nn.d3);
#else
  q.d3 = __subc_cc(q.d3, nn.d3);
  q.d4 = __subc   (q.d4, nn.d4);
#endif

/********** Step 5, Offset 2^0 (0*32 + 0) **********/
  MODBASECASE_NONZERO_ERROR(q.d5, 5, 5, 6);
  MODBASECASE_NONZERO_ERROR(q.d4, 5, 4, 7);

  qf= __uint2float_rn(q.d3);
  qf= qf * 4294967296.0f + __uint2float_rn(q.d2);
  qf= qf * 4294967296.0f + __uint2float_rn(q.d1);
  qf= qf * 4294967296.0f + __uint2float_rn(q.d0);
  
  qi=__float2uint_rz(qf*nf);

  MODBASECASE_QI_ERROR(1<<20, 5, qi, 8);

// nn = n * qi
  nn.d0 =                                 __umul32(n.d0, qi);
  nn.d1 = __add_cc (__umul32hi(n.d0, qi), __umul32(n.d1, qi));
#ifndef CHECKS_MODBASECASE  
  nn.d2 = __addc   (__umul32hi(n.d1, qi), __umul32(n.d2, qi));
#else
  nn.d2 = __addc_cc(__umul32hi(n.d1, qi), __umul32(n.d2, qi));
  nn.d3 = __addc   (__umul32hi(n.d2, qi),                  0);
#endif  

//  q = q - nn
  q.d0 = __sub_cc (q.d0, nn.d0);
  q.d1 = __subc_cc(q.d1, nn.d1);
#ifndef CHECKS_MODBASECASE
  q.d2 = __subc   (q.d2, nn.d2);
#else
  q.d2 = __subc_cc(q.d2, nn.d2);
  q.d3 = __subc   (q.d3, nn.d3);
#endif

  res->d0=q.d0;
  res->d1=q.d1;
  res->d2=q.d2;
  
  MODBASECASE_NONZERO_ERROR(q.d5, 6, 5, 9);
  MODBASECASE_NONZERO_ERROR(q.d4, 6, 4, 10);
  MODBASECASE_NONZERO_ERROR(q.d3, 6, 3, 11);

/*
qi is allways a little bit too small, this is OK for all steps except the last
one. Sometimes the result is a little bit bigger than n
*/
/*  if(cmp_ge_96(*res,n))
  {
    sub_96(&tmp96,*res,n);
    copy_96(res,tmp96);
  }*/
}


__global__ void
#ifdef SHORTCUT_75BIT
  #ifndef CHECKS_MODBASECASE
__launch_bounds__(THREADS_PER_BLOCK,2) mfaktc_75(unsigned int exp, int96 k, unsigned int *k_tab, int shiftcount, int192 b, unsigned int *RES)
  #else
__launch_bounds__(THREADS_PER_BLOCK,2) mfaktc_75(unsigned int exp, int96 k, unsigned int *k_tab, int shiftcount, int192 b, unsigned int *RES, unsigned int *modbasecase_debug)
  #endif
#else
  #ifndef CHECKS_MODBASECASE
__launch_bounds__(THREADS_PER_BLOCK,2) mfaktc_95(unsigned int exp, int96 k, unsigned int *k_tab, int shiftcount, int192 b, unsigned int *RES)
  #else
__launch_bounds__(THREADS_PER_BLOCK,2) mfaktc_95(unsigned int exp, int96 k, unsigned int *k_tab, int shiftcount, int192 b, unsigned int *RES, unsigned int *modbasecase_debug)
  #endif
#endif
/*
computes 2^exp mod f
shiftcount is used for precomputing without mod
a is precomputed on host ONCE. */
{
  int96 f;
  int96 a;
  int index = blockDim.x * blockIdx.x + threadIdx.x;
  float ff;

  create_FC96_mad(&f, exp, k, k_tab[index]);    // f = 2 * (k + k_tab[index]) * exp + 1

/*
ff = f as float, needed in mod_192_96().
Precalculated here since it is the same for all steps in the following loop */
  ff= __uint2float_rn(f.d2);
  ff= ff * 4294967296.0f + __uint2float_rn(f.d1);
  ff= ff * 4294967296.0f + __uint2float_rn(f.d0);

  ff=__int_as_float(0x3f7ffffb) / ff;	// just a little bit below 1.0f so we allways underestimate the quotient
        
#ifndef CHECKS_MODBASECASE
  mod_192_96(&a,b,f,ff);			// a = b mod f
#else
  mod_192_96(&a,b,f,ff,modbasecase_debug);	// a = b mod f
#endif
  exp<<= 32 - shiftcount;
  while(exp)
  {
#ifdef SHORTCUT_75BIT
    square_96_160(&b,a);			// b = a^2
#else
    square_96_192(&b,a);			// b = a^2
#endif
    if(exp&0x80000000)shl_192(&b);              // "optional multiply by 2" in Prime 95 documentation
#ifndef CHECKS_MODBASECASE
      mod_192_96(&a,b,f,ff);			// a = b mod f
#else
      mod_192_96(&a,b,f,ff,modbasecase_debug);	// a = b mod f
#endif
    exp<<=1;
  }

  if(cmp_ge_96(a,f))				// final adjustment in case a >= f
  {
    sub_96(&a,a,f);
  }

#if defined CHECKS_MODBASECASE && defined USE_DEVICE_PRINTF && __CUDA_ARCH__ >= FERMI
  if(cmp_ge_96(a,f))
  {
    printf("EEEEEK, final a is >= f\n");
  }
#endif

/* finally check if we found a factor and write the factor to RES[]
this kernel has a lower FC limit of 2^64 so we can use check_big_factor96() */
  check_factor96(f, a, RES);
}

#define TF_96BIT
#include "tf_common.cu"
#undef TF_96BIT

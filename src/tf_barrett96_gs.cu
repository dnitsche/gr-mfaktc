#include "hip/hip_runtime.h"
/*
This file is part of mfaktc.
Copyright (C) 2009, 2010, 2011, 2012  Oliver Weihe (o.weihe@t-online.de)

mfaktc is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

mfaktc is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with mfaktc.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>  

#include "params.h"
#include "my_types.h"
#include "compatibility.h"
#include "my_intrinsics.h"

#define NVCC_EXTERN
#include "sieve.h"
#include "timer.h"
#include "output.h"
#undef NVCC_EXTERN

#include "tf_debug.h"
#include "tf_96bit_base_math.cu"
#include "tf_96bit_helper.cu"

#undef DIV_160_96
#include "tf_barrett96_div.cu"
#define DIV_160_96
#include "tf_barrett96_div.cu"
#undef DIV_160_96

#include "tf_barrett96_core.cu"


// Inline to find the highest set bit in a word
// If no bit is set, CC 2.x returns 32, CC 1.x returns 31

__device__ static unsigned int ___clz (unsigned int a)
{
#if (__CUDA_ARCH__ >= FERMI) /* clz (count leading zeroes) is not available on CC 1.x devices */
	unsigned int r;
	asm("clz.b32 %0, %1;" : "=r" (r) : "r" (a));
	return r;
#else
	unsigned int r = 0;
	if ((a & 0xFFFF0000) == 0) r = 16, a <<= 16;
	if ((a & 0xFF000000) == 0) r += 8, a <<= 8;
	if ((a & 0xF0000000) == 0) r += 4, a <<= 4;
	if ((a & 0xC0000000) == 0) r += 2, a <<= 2;
	if ((a & 0x80000000) == 0) r += 1;
	return r;
#endif
}

// Inline to count the number of set bits in a word

__device__ static unsigned int ___popcnt (unsigned int a)
{
#if (__CUDA_ARCH__ >= FERMI) /* popc (population count) is not available on CC 1.x devices */
	unsigned int r;
	asm("popc.b32 %0, %1;" : "=r" (r) : "r" (a));
	return r;
#else
	a = (a&0x55555555) + ((a>> 1)&0x55555555);  // Generate sixteen 2-bit sums
	a = (a&0x33333333) + ((a>> 2)&0x33333333);  // Generate eight 3-bit sums
	a = (a&0x07070707) + ((a>> 4)&0x07070707);  // Generate four 4-bit sums
	a = (a&0x000F000F) + ((a>> 8)&0x000F000F);  // Generate two 5-bit sums
	a = (a&0x0000001F) + ((a>>16)&0x0000001F);  // Generate one 6-bit sum
	return a;
#endif
}


#if __CUDA_ARCH__ >= FERMI
  #define KERNEL_MIN_BLOCKS 2
#else
  #define KERNEL_MIN_BLOCKS 1
#endif

__global__ void
#ifndef CHECKS_MODBASECASE
__launch_bounds__(THREADS_PER_BLOCK, KERNEL_MIN_BLOCKS) mfaktc_barrett92_gs(unsigned int exp, int96 k_base, unsigned int *bit_array, unsigned int bits_to_process, int shiftcount, int192 b_preinit, unsigned int *RES, int bit_max64)
#else
__launch_bounds__(THREADS_PER_BLOCK, KERNEL_MIN_BLOCKS) mfaktc_barrett92_gs(unsigned int exp, int96 k_base, unsigned int *bit_array, unsigned int bits_to_process, int shiftcount, int192 b_preinit, unsigned int *RES, int bit_max64, unsigned int *modbasecase_debug)
#endif
/*
computes 2^exp mod f
shiftcount is used for precomputing without mod
a is precomputed on host ONCE.
bit_max64 is the number of bits in the factor (minus 64)
*/
{
  int96 f_base;
  int i, words_per_thread, initial_shifter_value, sieve_word, k_bit_base, total_bit_count;
  __shared__ volatile unsigned short bitcount[256];	// Each thread of our block puts bit-counts here
  extern __shared__ unsigned short smem[];		// Write bits to test here.  Launching program must estimate
							// how much shared memory to allocate based on number of primes sieved.

  // Get pointer to section of the bit_array this thread is processing.

  words_per_thread = bits_to_process / 8192;
  bit_array += blockIdx.x * bits_to_process / 32 + threadIdx.x * words_per_thread;

// Count number of bits set in this thread's word(s) from the bit_array

  bitcount[threadIdx.x] = 0;
  for (i = 0; i < words_per_thread; i++)
    bitcount[threadIdx.x] += ___popcnt(bit_array[i]);

// Create total count of bits set in block up to and including this threads popcnt.
// Kudos to Rocke Verser for the population counting code.
// CAUTION:  Following requires 256 threads per block

  // First five tallies remain within one warp.  Should be in lock-step.
  if (threadIdx.x & 1)        // If we are running on any thread 0bxxxxxxx1, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[threadIdx.x - 1];

  if (threadIdx.x & 2)        // If we are running on any thread 0bxxxxxx1x, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 2) | 1];

  if (threadIdx.x & 4)        // If we are running on any thread 0bxxxxx1xx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 4) | 3];

  if (threadIdx.x & 8)        // If we are running on any thread 0bxxxx1xxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 8) | 7];

  if (threadIdx.x & 16)       // If we are running on any thread 0bxxx1xxxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 16) | 15];

  // Further tallies are across warps.  Must synchronize
  __syncthreads();
  if (threadIdx.x  & 32)      // If we are running on any thread 0bxx1xxxxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 32) | 31];

  __syncthreads();
  if (threadIdx.x & 64)       // If we are running on any thread 0bx1xxxxxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 64) | 63];

  __syncthreads();
  if (threadIdx.x & 128)       // If we are running on any thread 0b1xxxxxxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[127];

  // At this point, bitcount[...] contains the total number of bits for the indexed
  // thread plus all lower-numbered threads.  I.e., bitcount[255] is the total count.

  __syncthreads();
  total_bit_count = bitcount[255];

//POSSIBLE OPTIMIZATION - bitcounts and smem could use the same memory space if we'd read bitcount into a register
// and sync threads before doing any writes to smem.

//POSSIBLE SANITY CHECK -- is there any way to test if total_bit_count exceeds the amount of shared memory allocated?

// Loop til this thread's section of the bit array is finished.

  sieve_word = *bit_array;
  k_bit_base = threadIdx.x * words_per_thread * 32;
  for (i = total_bit_count - bitcount[threadIdx.x]; ; i++) {
    int bit_to_test;

// Make sure we have a non-zero sieve word

    while (sieve_word == 0) {
      if (--words_per_thread == 0) break;
      sieve_word = *++bit_array;
      k_bit_base += 32;
    }

// Check if this thread has processed all its set bits

    if (sieve_word == 0) break;

// Find a bit to test in the sieve word

    bit_to_test = 31 - ___clz (sieve_word);
    sieve_word &= ~(1 << bit_to_test);

// Copy the k value to the shared memory array

    smem[i] = k_bit_base + bit_to_test;
  }

  __syncthreads();

// Here, all warps in our block have placed their candidates in shared memory.
// Now we can start TFing candidates.

// Init some stuff that will be used for all k's tested

  initial_shifter_value = exp << (32 - shiftcount);	// Initial shifter value

// Compute factor corresponding to first sieve bit in this block.

  // Compute base k value
  k_base.d0 = __add_cc (k_base.d0, __umul32  (blockIdx.x * bits_to_process, NUM_CLASSES));
  k_base.d1 = __addc   (k_base.d1, __umul32hi(blockIdx.x * bits_to_process, NUM_CLASSES)); /* k values are limited to 64 bits */

  // Compute k * exp
  f_base.d0 =                                      __umul32(k_base.d0, exp);
  f_base.d1 = __add_cc(__umul32hi(k_base.d0, exp), __umul32(k_base.d1, exp));
  f_base.d2 = __addc  (__umul32hi(k_base.d1, exp),                       0);

  // Compute f_base = 2 * k * exp + 1
  shl_96(&f_base);
  f_base.d0 = f_base.d0 + 1;

// Loop til the k values written to shared memory are exhausted

  for (i = threadIdx.x; i < total_bit_count; i += THREADS_PER_BLOCK) {
    int96 f;
    int k_delta;

// Get the (k - k_base) value to test

    k_delta = smem[i];

// Compute new f.  This is computed as f = f_base + 2 * (k - k_base) * exp.

    f.d0 = __add_cc (f_base.d0, __umul32(2 * k_delta * NUM_CLASSES, exp));
    f.d1 = __addc_cc(f_base.d1, __umul32hi(2 * k_delta * NUM_CLASSES, exp));
    f.d2 = __addc   (f_base.d2, 0);

    test_FC96_barrett92(f, b_preinit, initial_shifter_value, RES, bit_max64
#ifdef CHECKS_MODBASECASE
                        , modbasecase_debug
#endif
                        );
  }
}


__global__ void
#ifndef CHECKS_MODBASECASE
__launch_bounds__(THREADS_PER_BLOCK, KERNEL_MIN_BLOCKS) mfaktc_barrett88_gs(unsigned int exp, int96 k_base, unsigned int *bit_array, unsigned int bits_to_process, int shiftcount, int192 b_preinit, unsigned int *RES, int bit_max64)
#else
__launch_bounds__(THREADS_PER_BLOCK, KERNEL_MIN_BLOCKS) mfaktc_barrett88_gs(unsigned int exp, int96 k_base, unsigned int *bit_array, unsigned int bits_to_process, int shiftcount, int192 b_preinit, unsigned int *RES, int bit_max64, unsigned int *modbasecase_debug)
#endif
/*
computes 2^exp mod f
shiftcount is used for precomputing without mod
a is precomputed on host ONCE.
bit_max64 is the number of bits in the factor (minus 64)
*/
{
  int96 f_base;
  int i, words_per_thread, initial_shifter_value, sieve_word, k_bit_base, total_bit_count;
  __shared__ volatile unsigned short bitcount[256];	// Each thread of our block puts bit-counts here
  extern __shared__ unsigned short smem[];		// Write bits to test here.  Launching program must estimate
							// how much shared memory to allocate based on number of primes sieved.

  // Get pointer to section of the bit_array this thread is processing.

  words_per_thread = bits_to_process / 8192;
  bit_array += blockIdx.x * bits_to_process / 32 + threadIdx.x * words_per_thread;

// Count number of bits set in this thread's word(s) from the bit_array

  bitcount[threadIdx.x] = 0;
  for (i = 0; i < words_per_thread; i++)
    bitcount[threadIdx.x] += ___popcnt(bit_array[i]);

// Create total count of bits set in block up to and including this threads popcnt.
// Kudos to Rocke Verser for the population counting code.
// CAUTION:  Following requires 256 threads per block

  // First five tallies remain within one warp.  Should be in lock-step.
  if (threadIdx.x & 1)        // If we are running on any thread 0bxxxxxxx1, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[threadIdx.x - 1];

  if (threadIdx.x & 2)        // If we are running on any thread 0bxxxxxx1x, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 2) | 1];

  if (threadIdx.x & 4)        // If we are running on any thread 0bxxxxx1xx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 4) | 3];

  if (threadIdx.x & 8)        // If we are running on any thread 0bxxxx1xxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 8) | 7];

  if (threadIdx.x & 16)       // If we are running on any thread 0bxxx1xxxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 16) | 15];

  // Further tallies are across warps.  Must synchronize
  __syncthreads();
  if (threadIdx.x  & 32)      // If we are running on any thread 0bxx1xxxxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 32) | 31];

  __syncthreads();
  if (threadIdx.x & 64)       // If we are running on any thread 0bx1xxxxxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 64) | 63];

  __syncthreads();
  if (threadIdx.x & 128)       // If we are running on any thread 0b1xxxxxxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[127];

  // At this point, bitcount[...] contains the total number of bits for the indexed
  // thread plus all lower-numbered threads.  I.e., bitcount[255] is the total count.

  __syncthreads();
  total_bit_count = bitcount[255];

//POSSIBLE OPTIMIZATION - bitcounts and smem could use the same memory space if we'd read bitcount into a register
// and sync threads before doing any writes to smem.

//POSSIBLE SANITY CHECK -- is there any way to test if total_bit_count exceeds the amount of shared memory allocated?

// Loop til this thread's section of the bit array is finished.

  sieve_word = *bit_array;
  k_bit_base = threadIdx.x * words_per_thread * 32;
  for (i = total_bit_count - bitcount[threadIdx.x]; ; i++) {
    int bit_to_test;

// Make sure we have a non-zero sieve word

    while (sieve_word == 0) {
      if (--words_per_thread == 0) break;
      sieve_word = *++bit_array;
      k_bit_base += 32;
    }

// Check if this thread has processed all its set bits

    if (sieve_word == 0) break;

// Find a bit to test in the sieve word

    bit_to_test = 31 - ___clz (sieve_word);
    sieve_word &= ~(1 << bit_to_test);

// Copy the k value to the shared memory array

    smem[i] = k_bit_base + bit_to_test;
  }

  __syncthreads();

// Here, all warps in our block have placed their candidates in shared memory.
// Now we can start TFing candidates.

// Init some stuff that will be used for all k's tested

  initial_shifter_value = exp << (32 - shiftcount);	// Initial shifter value

// Compute factor corresponding to first sieve bit in this block.

  // Compute base k value
  k_base.d0 = __add_cc (k_base.d0, __umul32  (blockIdx.x * bits_to_process, NUM_CLASSES));
  k_base.d1 = __addc   (k_base.d1, __umul32hi(blockIdx.x * bits_to_process, NUM_CLASSES)); /* k values are limited to 64 bits */

  // Compute k * exp
  f_base.d0 =                                      __umul32(k_base.d0, exp);
  f_base.d1 = __add_cc(__umul32hi(k_base.d0, exp), __umul32(k_base.d1, exp));
  f_base.d2 = __addc  (__umul32hi(k_base.d1, exp),                       0);

  // Compute f_base = 2 * k * exp + 1
  shl_96(&f_base);
  f_base.d0 = f_base.d0 + 1;

// Loop til the k values written to shared memory are exhausted

  for (i = threadIdx.x; i < total_bit_count; i += THREADS_PER_BLOCK) {
    int96 f;
    int k_delta;

// Get the (k - k_base) value to test

    k_delta = smem[i];

// Compute new f.  This is computed as f = f_base + 2 * (k - k_base) * exp.

    f.d0 = __add_cc (f_base.d0, __umul32(2 * k_delta * NUM_CLASSES, exp));
    f.d1 = __addc_cc(f_base.d1, __umul32hi(2 * k_delta * NUM_CLASSES, exp));
    f.d2 = __addc   (f_base.d2, 0);

    test_FC96_barrett88(f, b_preinit, initial_shifter_value, RES, bit_max64
#ifdef CHECKS_MODBASECASE
                        , modbasecase_debug
#endif
                        );
  }
}


__global__ void
#ifndef CHECKS_MODBASECASE
__launch_bounds__(THREADS_PER_BLOCK, KERNEL_MIN_BLOCKS) mfaktc_barrett87_gs(unsigned int exp, int96 k_base, unsigned int *bit_array, unsigned int bits_to_process, int shiftcount, int192 b_preinit, unsigned int *RES, int bit_max64)
#else
__launch_bounds__(THREADS_PER_BLOCK, KERNEL_MIN_BLOCKS) mfaktc_barrett87_gs(unsigned int exp, int96 k_base, unsigned int *bit_array, unsigned int bits_to_process, int shiftcount, int192 b_preinit, unsigned int *RES, int bit_max64, unsigned int *modbasecase_debug)
#endif
/*
computes 2^exp mod f
shiftcount is used for precomputing without mod
a is precomputed on host ONCE.
bit_max64 is the number of bits in the factor (minus 64)
*/
{
  int96 f_base;
  int i, words_per_thread, initial_shifter_value, sieve_word, k_bit_base, total_bit_count;
  __shared__ volatile unsigned short bitcount[256];	// Each thread of our block puts bit-counts here
  extern __shared__ unsigned short smem[];		// Write bits to test here.  Launching program must estimate
							// how much shared memory to allocate based on number of primes sieved.

  // Get pointer to section of the bit_array this thread is processing.

  words_per_thread = bits_to_process / 8192;
  bit_array += blockIdx.x * bits_to_process / 32 + threadIdx.x * words_per_thread;

// Count number of bits set in this thread's word(s) from the bit_array

  bitcount[threadIdx.x] = 0;
  for (i = 0; i < words_per_thread; i++)
    bitcount[threadIdx.x] += ___popcnt(bit_array[i]);

// Create total count of bits set in block up to and including this threads popcnt.
// Kudos to Rocke Verser for the population counting code.
// CAUTION:  Following requires 256 threads per block

  // First five tallies remain within one warp.  Should be in lock-step.
  if (threadIdx.x & 1)        // If we are running on any thread 0bxxxxxxx1, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[threadIdx.x - 1];

  if (threadIdx.x & 2)        // If we are running on any thread 0bxxxxxx1x, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 2) | 1];

  if (threadIdx.x & 4)        // If we are running on any thread 0bxxxxx1xx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 4) | 3];

  if (threadIdx.x & 8)        // If we are running on any thread 0bxxxx1xxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 8) | 7];

  if (threadIdx.x & 16)       // If we are running on any thread 0bxxx1xxxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 16) | 15];

  // Further tallies are across warps.  Must synchronize
  __syncthreads();
  if (threadIdx.x  & 32)      // If we are running on any thread 0bxx1xxxxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 32) | 31];

  __syncthreads();
  if (threadIdx.x & 64)       // If we are running on any thread 0bx1xxxxxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 64) | 63];

  __syncthreads();
  if (threadIdx.x & 128)       // If we are running on any thread 0b1xxxxxxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[127];

  // At this point, bitcount[...] contains the total number of bits for the indexed
  // thread plus all lower-numbered threads.  I.e., bitcount[255] is the total count.

  __syncthreads();
  total_bit_count = bitcount[255];

//POSSIBLE OPTIMIZATION - bitcounts and smem could use the same memory space if we'd read bitcount into a register
// and sync threads before doing any writes to smem.

//POSSIBLE SANITY CHECK -- is there any way to test if total_bit_count exceeds the amount of shared memory allocated?

// Loop til this thread's section of the bit array is finished.

  sieve_word = *bit_array;
  k_bit_base = threadIdx.x * words_per_thread * 32;
  for (i = total_bit_count - bitcount[threadIdx.x]; ; i++) {
    int bit_to_test;

// Make sure we have a non-zero sieve word

    while (sieve_word == 0) {
      if (--words_per_thread == 0) break;
      sieve_word = *++bit_array;
      k_bit_base += 32;
    }

// Check if this thread has processed all its set bits

    if (sieve_word == 0) break;

// Find a bit to test in the sieve word

    bit_to_test = 31 - ___clz (sieve_word);
    sieve_word &= ~(1 << bit_to_test);

// Copy the k value to the shared memory array

    smem[i] = k_bit_base + bit_to_test;
  }

  __syncthreads();

// Here, all warps in our block have placed their candidates in shared memory.
// Now we can start TFing candidates.

// Init some stuff that will be used for all k's tested

  initial_shifter_value = exp << (32 - shiftcount);	// Initial shifter value

// Compute factor corresponding to first sieve bit in this block.

  // Compute base k value
  k_base.d0 = __add_cc (k_base.d0, __umul32  (blockIdx.x * bits_to_process, NUM_CLASSES));
  k_base.d1 = __addc   (k_base.d1, __umul32hi(blockIdx.x * bits_to_process, NUM_CLASSES)); /* k values are limited to 64 bits */

  // Compute k * exp
  f_base.d0 =                                      __umul32(k_base.d0, exp);
  f_base.d1 = __add_cc(__umul32hi(k_base.d0, exp), __umul32(k_base.d1, exp));
  f_base.d2 = __addc  (__umul32hi(k_base.d1, exp),                       0);

  // Compute f_base = 2 * k * exp + 1
  shl_96(&f_base);
  f_base.d0 = f_base.d0 + 1;

// Loop til the k values written to shared memory are exhausted

  for (i = threadIdx.x; i < total_bit_count; i += THREADS_PER_BLOCK) {
    int96 f;
    int k_delta;

// Get the (k - k_base) value to test

    k_delta = smem[i];

// Compute new f.  This is computed as f = f_base + 2 * (k - k_base) * exp.

    f.d0 = __add_cc (f_base.d0, __umul32(2 * k_delta * NUM_CLASSES, exp));
    f.d1 = __addc_cc(f_base.d1, __umul32hi(2 * k_delta * NUM_CLASSES, exp));
    f.d2 = __addc   (f_base.d2, 0);

    test_FC96_barrett87(f, b_preinit, initial_shifter_value, RES, bit_max64
#ifdef CHECKS_MODBASECASE
                        , modbasecase_debug
#endif
                        );
  }
}


__global__ void
#ifndef CHECKS_MODBASECASE
__launch_bounds__(THREADS_PER_BLOCK, KERNEL_MIN_BLOCKS) mfaktc_barrett79_gs(unsigned int exp, int96 k_base, unsigned int *bit_array, unsigned int bits_to_process, int shiftcount, int192 b_preinit, unsigned int *RES)
#else
__launch_bounds__(THREADS_PER_BLOCK, KERNEL_MIN_BLOCKS) mfaktc_barrett79_gs(unsigned int exp, int96 k_base, unsigned int *bit_array, unsigned int bits_to_process, int shiftcount, int192 b_preinit, unsigned int *RES, int bit_max64, unsigned int *modbasecase_debug)
#endif
/*
computes 2^exp mod f
shiftcount is used for precomputing without mod
a is precomputed on host ONCE.
*/
{
  int96 f_base;
  int i, words_per_thread, initial_shifter_value, sieve_word, k_bit_base, total_bit_count;
  __shared__ volatile unsigned short bitcount[256];	// Each thread of our block puts bit-counts here
  extern __shared__ unsigned short smem[];		// Write bits to test here.  Launching program must estimate
							// how much shared memory to allocate based on number of primes sieved.

  // Get pointer to section of the bit_array this thread is processing.

  words_per_thread = bits_to_process / 8192;
  bit_array += blockIdx.x * bits_to_process / 32 + threadIdx.x * words_per_thread;

// Count number of bits set in this thread's word(s) from the bit_array

  bitcount[threadIdx.x] = 0;
  for (i = 0; i < words_per_thread; i++)
    bitcount[threadIdx.x] += ___popcnt(bit_array[i]);

// Create total count of bits set in block up to and including this threads popcnt.
// Kudos to Rocke Verser for the population counting code.
// CAUTION:  Following requires 256 threads per block

  // First five tallies remain within one warp.  Should be in lock-step.
  if (threadIdx.x & 1)        // If we are running on any thread 0bxxxxxxx1, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[threadIdx.x - 1];

  if (threadIdx.x & 2)        // If we are running on any thread 0bxxxxxx1x, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 2) | 1];

  if (threadIdx.x & 4)        // If we are running on any thread 0bxxxxx1xx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 4) | 3];

  if (threadIdx.x & 8)        // If we are running on any thread 0bxxxx1xxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 8) | 7];

  if (threadIdx.x & 16)       // If we are running on any thread 0bxxx1xxxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 16) | 15];

  // Further tallies are across warps.  Must synchronize
  __syncthreads();
  if (threadIdx.x  & 32)      // If we are running on any thread 0bxx1xxxxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 32) | 31];

  __syncthreads();
  if (threadIdx.x & 64)       // If we are running on any thread 0bx1xxxxxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 64) | 63];

  __syncthreads();
  if (threadIdx.x & 128)       // If we are running on any thread 0b1xxxxxxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[127];

  // At this point, bitcount[...] contains the total number of bits for the indexed
  // thread plus all lower-numbered threads.  I.e., bitcount[255] is the total count.

  __syncthreads();
  total_bit_count = bitcount[255];

//POSSIBLE OPTIMIZATION - bitcounts and smem could use the same memory space if we'd read bitcount into a register
// and sync threads before doing any writes to smem.

//POSSIBLE SANITY CHECK -- is there any way to test if total_bit_count exceeds the amount of shared memory allocated?

// Loop til this thread's section of the bit array is finished.

  sieve_word = *bit_array;
  k_bit_base = threadIdx.x * words_per_thread * 32;
  for (i = total_bit_count - bitcount[threadIdx.x]; ; i++) {
    int bit_to_test;

// Make sure we have a non-zero sieve word

    while (sieve_word == 0) {
      if (--words_per_thread == 0) break;
      sieve_word = *++bit_array;
      k_bit_base += 32;
    }

// Check if this thread has processed all its set bits

    if (sieve_word == 0) break;

// Find a bit to test in the sieve word

    bit_to_test = 31 - ___clz (sieve_word);
    sieve_word &= ~(1 << bit_to_test);

// Copy the k value to the shared memory array

    smem[i] = k_bit_base + bit_to_test;
  }

  __syncthreads();

// Here, all warps in our block have placed their candidates in shared memory.
// Now we can start TFing candidates.

// Init some stuff that will be used for all k's tested

  initial_shifter_value = exp << (32 - shiftcount);	// Initial shifter value

// Compute factor corresponding to first sieve bit in this block.

  // Compute base k value
  k_base.d0 = __add_cc (k_base.d0, __umul32  (blockIdx.x * bits_to_process, NUM_CLASSES));
  k_base.d1 = __addc   (k_base.d1, __umul32hi(blockIdx.x * bits_to_process, NUM_CLASSES)); /* k values are limited to 64 bits */

  // Compute k * exp
  f_base.d0 =                                      __umul32(k_base.d0, exp);
  f_base.d1 = __add_cc(__umul32hi(k_base.d0, exp), __umul32(k_base.d1, exp));
  f_base.d2 = __addc  (__umul32hi(k_base.d1, exp),                       0);

  // Compute f_base = 2 * k * exp + 1
  shl_96(&f_base);
  f_base.d0 = f_base.d0 + 1;

// Loop til the k values written to shared memory are exhausted

  for (i = threadIdx.x; i < total_bit_count; i += THREADS_PER_BLOCK) {
    int96 f;
    int k_delta;

// Get the (k - k_base) value to test

    k_delta = smem[i];

// Compute new f.  This is computed as f = f_base + 2 * (k - k_base) * exp.

    f.d0 = __add_cc (f_base.d0, __umul32(2 * k_delta * NUM_CLASSES, exp));
    f.d1 = __addc_cc(f_base.d1, __umul32hi(2 * k_delta * NUM_CLASSES, exp));
    f.d2 = __addc   (f_base.d2, 0);

    test_FC96_barrett79(f, b_preinit, initial_shifter_value, RES
#ifdef CHECKS_MODBASECASE
                        , bit_max64, modbasecase_debug
#endif
                        );
  }
}


__global__ void
#ifndef CHECKS_MODBASECASE
__launch_bounds__(THREADS_PER_BLOCK, KERNEL_MIN_BLOCKS) mfaktc_barrett77_gs(unsigned int exp, int96 k_base, unsigned int *bit_array, unsigned int bits_to_process, int shiftcount, int192 b_preinit, unsigned int *RES)
#else
__launch_bounds__(THREADS_PER_BLOCK, KERNEL_MIN_BLOCKS) mfaktc_barrett77_gs(unsigned int exp, int96 k_base, unsigned int *bit_array, unsigned int bits_to_process, int shiftcount, int192 b_preinit, unsigned int *RES, int bit_max64, unsigned int *modbasecase_debug)
#endif
/*
computes 2^exp mod f
shiftcount is used for precomputing without mod
a is precomputed on host ONCE.
*/
{
  int96 f_base;
  int i, words_per_thread, initial_shifter_value, sieve_word, k_bit_base, total_bit_count;
  __shared__ volatile unsigned short bitcount[256];	// Each thread of our block puts bit-counts here
  extern __shared__ unsigned short smem[];		// Write bits to test here.  Launching program must estimate
							// how much shared memory to allocate based on number of primes sieved.

  // Get pointer to section of the bit_array this thread is processing.

  words_per_thread = bits_to_process / 8192;
  bit_array += blockIdx.x * bits_to_process / 32 + threadIdx.x * words_per_thread;

// Count number of bits set in this thread's word(s) from the bit_array

  bitcount[threadIdx.x] = 0;
  for (i = 0; i < words_per_thread; i++)
    bitcount[threadIdx.x] += ___popcnt(bit_array[i]);

// Create total count of bits set in block up to and including this threads popcnt.
// Kudos to Rocke Verser for the population counting code.
// CAUTION:  Following requires 256 threads per block

  // First five tallies remain within one warp.  Should be in lock-step.
  if (threadIdx.x & 1)        // If we are running on any thread 0bxxxxxxx1, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[threadIdx.x - 1];

  if (threadIdx.x & 2)        // If we are running on any thread 0bxxxxxx1x, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 2) | 1];

  if (threadIdx.x & 4)        // If we are running on any thread 0bxxxxx1xx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 4) | 3];

  if (threadIdx.x & 8)        // If we are running on any thread 0bxxxx1xxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 8) | 7];

  if (threadIdx.x & 16)       // If we are running on any thread 0bxxx1xxxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 16) | 15];

  // Further tallies are across warps.  Must synchronize
  __syncthreads();
  if (threadIdx.x  & 32)      // If we are running on any thread 0bxx1xxxxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 32) | 31];

  __syncthreads();
  if (threadIdx.x & 64)       // If we are running on any thread 0bx1xxxxxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 64) | 63];

  __syncthreads();
  if (threadIdx.x & 128)       // If we are running on any thread 0b1xxxxxxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[127];

  // At this point, bitcount[...] contains the total number of bits for the indexed
  // thread plus all lower-numbered threads.  I.e., bitcount[255] is the total count.

  __syncthreads();
  total_bit_count = bitcount[255];

//POSSIBLE OPTIMIZATION - bitcounts and smem could use the same memory space if we'd read bitcount into a register
// and sync threads before doing any writes to smem.

//POSSIBLE SANITY CHECK -- is there any way to test if total_bit_count exceeds the amount of shared memory allocated?

// Loop til this thread's section of the bit array is finished.

  sieve_word = *bit_array;
  k_bit_base = threadIdx.x * words_per_thread * 32;
  for (i = total_bit_count - bitcount[threadIdx.x]; ; i++) {
    int bit_to_test;

// Make sure we have a non-zero sieve word

    while (sieve_word == 0) {
      if (--words_per_thread == 0) break;
      sieve_word = *++bit_array;
      k_bit_base += 32;
    }

// Check if this thread has processed all its set bits

    if (sieve_word == 0) break;

// Find a bit to test in the sieve word

    bit_to_test = 31 - ___clz (sieve_word);
    sieve_word &= ~(1 << bit_to_test);

// Copy the k value to the shared memory array

    smem[i] = k_bit_base + bit_to_test;
  }

  __syncthreads();

// Here, all warps in our block have placed their candidates in shared memory.
// Now we can start TFing candidates.

// Init some stuff that will be used for all k's tested

  initial_shifter_value = exp << (32 - shiftcount);	// Initial shifter value

// Compute factor corresponding to first sieve bit in this block.

  // Compute base k value
  k_base.d0 = __add_cc (k_base.d0, __umul32  (blockIdx.x * bits_to_process, NUM_CLASSES));
  k_base.d1 = __addc   (k_base.d1, __umul32hi(blockIdx.x * bits_to_process, NUM_CLASSES)); /* k values are limited to 64 bits */

  // Compute k * exp
  f_base.d0 =                                      __umul32(k_base.d0, exp);
  f_base.d1 = __add_cc(__umul32hi(k_base.d0, exp), __umul32(k_base.d1, exp));
  f_base.d2 = __addc  (__umul32hi(k_base.d1, exp),                       0);

  // Compute f_base = 2 * k * exp + 1
  shl_96(&f_base);
  f_base.d0 = f_base.d0 + 1;

// Loop til the k values written to shared memory are exhausted

  for (i = threadIdx.x; i < total_bit_count; i += THREADS_PER_BLOCK) {
    int96 f;
    int k_delta;

// Get the (k - k_base) value to test

    k_delta = smem[i];

// Compute new f.  This is computed as f = f_base + 2 * (k - k_base) * exp.

    f.d0 = __add_cc (f_base.d0, __umul32(2 * k_delta * NUM_CLASSES, exp));
    f.d1 = __addc_cc(f_base.d1, __umul32hi(2 * k_delta * NUM_CLASSES, exp));
    f.d2 = __addc   (f_base.d2, 0);

    test_FC96_barrett77(f, b_preinit, initial_shifter_value, RES
#ifdef CHECKS_MODBASECASE
                        , bit_max64, modbasecase_debug
#endif
                        );
  }
}


__global__ void
#ifndef CHECKS_MODBASECASE
__launch_bounds__(THREADS_PER_BLOCK, KERNEL_MIN_BLOCKS) mfaktc_barrett76_gs(unsigned int exp, int96 k_base, unsigned int *bit_array, unsigned int bits_to_process, int shiftcount, int192 b_preinit, unsigned int *RES)
#else
__launch_bounds__(THREADS_PER_BLOCK, KERNEL_MIN_BLOCKS) mfaktc_barrett76_gs(unsigned int exp, int96 k_base, unsigned int *bit_array, unsigned int bits_to_process, int shiftcount, int192 b_preinit, unsigned int *RES, int bit_max64, unsigned int *modbasecase_debug)
#endif
/*
computes 2^exp mod f
shiftcount is used for precomputing without mod
a is precomputed on host ONCE.
*/
{
  int96 f_base;
  int i, words_per_thread, initial_shifter_value, sieve_word, k_bit_base, total_bit_count;
  __shared__ volatile unsigned short bitcount[256];	// Each thread of our block puts bit-counts here
  extern __shared__ unsigned short smem[];		// Write bits to test here.  Launching program must estimate
							// how much shared memory to allocate based on number of primes sieved.

  // Get pointer to section of the bit_array this thread is processing.

  words_per_thread = bits_to_process / 8192;
  bit_array += blockIdx.x * bits_to_process / 32 + threadIdx.x * words_per_thread;

// Count number of bits set in this thread's word(s) from the bit_array

  bitcount[threadIdx.x] = 0;
  for (i = 0; i < words_per_thread; i++)
    bitcount[threadIdx.x] += ___popcnt(bit_array[i]);

// Create total count of bits set in block up to and including this threads popcnt.
// Kudos to Rocke Verser for the population counting code.
// CAUTION:  Following requires 256 threads per block

  // First five tallies remain within one warp.  Should be in lock-step.
  if (threadIdx.x & 1)        // If we are running on any thread 0bxxxxxxx1, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[threadIdx.x - 1];

  if (threadIdx.x & 2)        // If we are running on any thread 0bxxxxxx1x, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 2) | 1];

  if (threadIdx.x & 4)        // If we are running on any thread 0bxxxxx1xx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 4) | 3];

  if (threadIdx.x & 8)        // If we are running on any thread 0bxxxx1xxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 8) | 7];

  if (threadIdx.x & 16)       // If we are running on any thread 0bxxx1xxxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 16) | 15];

  // Further tallies are across warps.  Must synchronize
  __syncthreads();
  if (threadIdx.x  & 32)      // If we are running on any thread 0bxx1xxxxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 32) | 31];

  __syncthreads();
  if (threadIdx.x & 64)       // If we are running on any thread 0bx1xxxxxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[(threadIdx.x - 64) | 63];

  __syncthreads();
  if (threadIdx.x & 128)       // If we are running on any thread 0b1xxxxxxx, tally neighbor's count.
    bitcount[threadIdx.x] += bitcount[127];

  // At this point, bitcount[...] contains the total number of bits for the indexed
  // thread plus all lower-numbered threads.  I.e., bitcount[255] is the total count.

  __syncthreads();
  total_bit_count = bitcount[255];

//POSSIBLE OPTIMIZATION - bitcounts and smem could use the same memory space if we'd read bitcount into a register
// and sync threads before doing any writes to smem.

//POSSIBLE SANITY CHECK -- is there any way to test if total_bit_count exceeds the amount of shared memory allocated?

// Loop til this thread's section of the bit array is finished.

  sieve_word = *bit_array;
  k_bit_base = threadIdx.x * words_per_thread * 32;
  for (i = total_bit_count - bitcount[threadIdx.x]; ; i++) {
    int bit_to_test;

// Make sure we have a non-zero sieve word

    while (sieve_word == 0) {
      if (--words_per_thread == 0) break;
      sieve_word = *++bit_array;
      k_bit_base += 32;
    }

// Check if this thread has processed all its set bits

    if (sieve_word == 0) break;

// Find a bit to test in the sieve word

    bit_to_test = 31 - ___clz (sieve_word);
    sieve_word &= ~(1 << bit_to_test);

// Copy the k value to the shared memory array

    smem[i] = k_bit_base + bit_to_test;
  }

  __syncthreads();

// Here, all warps in our block have placed their candidates in shared memory.
// Now we can start TFing candidates.

// Init some stuff that will be used for all k's tested

  initial_shifter_value = exp << (32 - shiftcount);	// Initial shifter value

// Compute factor corresponding to first sieve bit in this block.

  // Compute base k value
  k_base.d0 = __add_cc (k_base.d0, __umul32  (blockIdx.x * bits_to_process, NUM_CLASSES));
  k_base.d1 = __addc   (k_base.d1, __umul32hi(blockIdx.x * bits_to_process, NUM_CLASSES)); /* k values are limited to 64 bits */

  // Compute k * exp
  f_base.d0 =                                      __umul32(k_base.d0, exp);
  f_base.d1 = __add_cc(__umul32hi(k_base.d0, exp), __umul32(k_base.d1, exp));
  f_base.d2 = __addc  (__umul32hi(k_base.d1, exp),                       0);

  // Compute f_base = 2 * k * exp + 1
  shl_96(&f_base);
  f_base.d0 = f_base.d0 + 1;

// Loop til the k values written to shared memory are exhausted

  for (i = threadIdx.x; i < total_bit_count; i += THREADS_PER_BLOCK) {
    int96 f;
    int k_delta;

// Get the (k - k_base) value to test

    k_delta = smem[i];

// Compute new f.  This is computed as f = f_base + 2 * (k - k_base) * exp.

    f.d0 = __add_cc (f_base.d0, __umul32(2 * k_delta * NUM_CLASSES, exp));
    f.d1 = __addc_cc(f_base.d1, __umul32hi(2 * k_delta * NUM_CLASSES, exp));
    f.d2 = __addc   (f_base.d2, 0);

    test_FC96_barrett76(f, b_preinit, initial_shifter_value, RES
#ifdef CHECKS_MODBASECASE
                        , bit_max64, modbasecase_debug
#endif
                        );
  }
}


#define TF_BARRETT

#define TF_BARRETT_92BIT_GS
#include "tf_common_gs.cu"
#undef TF_BARRETT_92BIT_GS

#define TF_BARRETT_88BIT_GS
#include "tf_common_gs.cu"
#undef TF_BARRETT_88BIT_GS

#define TF_BARRETT_87BIT_GS
#include "tf_common_gs.cu"
#undef TF_BARRETT_87BIT_GS

#define TF_BARRETT_79BIT_GS
#include "tf_common_gs.cu"
#undef TF_BARRETT_79BIT_GS

#define TF_BARRETT_77BIT_GS
#include "tf_common_gs.cu"
#undef TF_BARRETT_77BIT_GS

#define TF_BARRETT_76BIT_GS
#include "tf_common_gs.cu"
#undef TF_BARRETT_76BIT_GS

#undef TF_BARRETT

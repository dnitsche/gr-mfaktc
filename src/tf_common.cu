#include "hip/hip_runtime.h"
/*
This file is part of mfaktc.
Copyright (C) 2009, 2010, 2011, 2012, 2013, 2014  Oliver Weihe (o.weihe@t-online.de)

mfaktc is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

mfaktc is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with mfaktc.  If not, see <http://www.gnu.org/licenses/>.
*/


#ifdef SHORTCUT_64BIT
extern "C" __host__ int tf_class_64(unsigned long long int k_min, unsigned long long int k_max, mystuff_t *mystuff)
#define MFAKTC_FUNC mfaktc_64
#elif defined (SHORTCUT_75BIT)
extern "C" __host__ int tf_class_75(unsigned long long int k_min, unsigned long long int k_max, mystuff_t *mystuff)
#define MFAKTC_FUNC mfaktc_75
#else
extern "C" __host__ int tf_class_95(unsigned long long int k_min, unsigned long long int k_max, mystuff_t *mystuff)
#define MFAKTC_FUNC mfaktc_95
#endif
{
  size_t size = mystuff->threads_per_grid * sizeof(int);
  int i, index = 0, stream;
  hipError_t cuda_ret;
  timeval timer;
  timeval timer2;
  unsigned long long int twait = 0;
  int96 factor,k_base;
  int192 b_preinit;
  int shiftcount, logb, maxlogb, count = 0; // logarithm to base ´base´
  unsigned long long int k_diff;
  char string[50];
  int factorsfound = 0;
  bool is_base_negative = mystuff->base < 0;
  unsigned int abs_base = is_base_negative ? (unsigned int) -mystuff->base : (unsigned int) mystuff->base;

  int h_ktab_index = 0;
  int h_ktab_cpu[CPU_STREAMS_MAX];			// the set of h_ktab[N]s currently ownt by CPU
							// 0 <= N < h_ktab_index: these h_ktab[]s are preprocessed
                                                        // h_ktab_index <= N < mystuff.cpu_streams: these h_ktab[]s are NOT preprocessed
  int h_ktab_inuse[NUM_STREAMS_MAX];			// h_ktab_inuse[N] contains the number of h_ktab[] currently used by stream N
  unsigned long long int k_min_grid[CPU_STREAMS_MAX];	// k_min_grid[N] contains the k_min for h_ktab[h_ktab_cpu[N]], only valid for preprocessed h_ktab[]s

  timer_init(&timer);

  int threadsPerBlock = THREADS_PER_BLOCK;
  int blocksPerGrid = (mystuff->threads_per_grid + threadsPerBlock - 1) / threadsPerBlock;

  unsigned int delay = 1000;

  for(i=0; i<mystuff->num_streams; i++)h_ktab_inuse[i] = i;
  for(i=0; i<mystuff->cpu_streams; i++)h_ktab_cpu[i] = i + mystuff->num_streams;
  for(i=0; i<mystuff->cpu_streams; i++)k_min_grid[i] = 0;
  h_ktab_index = 0;

  shiftcount=0;
  // shiftcount = (int)log2(exponent), how many bits are there to process
  while((1ULL<<shiftcount) < (unsigned long long int)mystuff->exponent)shiftcount++;
//  printf("\n\nshiftcount = %d\n",shiftcount);
  shiftcount-=1;logb=1;
  maxlogb = shiftcount-3;
  // TODO: find maximum working numbers
#if defined (SHORTCUT_75BIT) || defined (SHORTCUT_64BIT)
  if (maxlogb>16) maxlogb=16; // maximum preprocessing which is possible for 64 bit
#else
  if (maxlogb>20) maxlogb=20; // maximum preprocessing which is possible
#endif
  if (abs_base<=10)
  {
    while(logb<maxlogb || 10*logb<mystuff->bit_min*3)	// how much preprocessing is possible
    {
      shiftcount--;
      logb<<=1; // log(x^2)
      if(mystuff->exponent&(1<<(shiftcount)))logb++; // optional mul with abs_base
    }
  }
//  printf("shiftcount = %d\n",shiftcount);
//  printf("logb = %d\n",logb);
  b_preinit.d5=0;b_preinit.d4=0;b_preinit.d3=0;b_preinit.d2=0;b_preinit.d1=0;b_preinit.d0=1;
// just calculate abs_base^logb
#ifdef SHORTCUT_64BIT
  for(i=0; i<logb; i++) mul64(&b_preinit, b_preinit, abs_base);
#elif defined (SHORTCUT_75BIT)
  for(i=0; i<logb; i++) mul75(&b_preinit, b_preinit, abs_base);
#else
  for(i=0; i<logb; i++) mul96(&b_preinit, b_preinit, abs_base);
#endif

/* set result array to 0 */
  hipMemsetAsync(mystuff->d_RES, 0, 1*sizeof(unsigned int)); //first int of result array contains the number of factors found

#ifdef DEBUG_GPU_MATH
  hipMemset(mystuff->d_modbasecase_debug, 0, 32*sizeof(int));
#endif

  timer_init(&timer2);
  while((k_min <= k_max) || (h_ktab_index > 0))
  {
/* preprocessing: calculate a ktab (factor table) */
    if((k_min <= k_max) && (h_ktab_index < mystuff->cpu_streams))	// if we have an empty h_ktab we can preprocess another one
    {
      delay = 1000;
      index = h_ktab_cpu[h_ktab_index];

      if(count > mystuff->num_streams)
      {
        twait+=timer_diff(&timer2);
      }
#ifdef DEBUG_STREAM_SCHEDULE
      printf(" STREAM_SCHEDULE: preprocessing on h_ktab[%d] (count = %d)\n", index, count);
#endif

      sieve_candidates(mystuff->threads_per_grid, mystuff->h_ktab[index], mystuff->sieve_primes);
      k_diff=mystuff->h_ktab[index][mystuff->threads_per_grid-1]+1;
      k_diff*=NUM_CLASSES;				/* NUM_CLASSES because classes are mod NUM_CLASSES */

      k_min_grid[h_ktab_index] = k_min;
      h_ktab_index++;

      count++;
      k_min += (unsigned long long int)k_diff;
      timer_init(&timer2);
    }
    else if(mystuff->allowsleep == 1)
    {
      /* no unused h_ktab for preprocessing.
      This usually means that
      a) all GPU streams are busy
      and
      b) we've preprocessed all available CPU streams
      so let's sleep for some time instead of running a busy loop on hipStreamQuery() */
      my_usleep(delay);

      delay = delay * 3 / 2;
      if(delay > 500000) delay = 500000;
    }


/* try upload ktab and start the calcualtion of a preprocessed dataset on the device */
    stream = 0;
    while((stream < mystuff->num_streams) && (h_ktab_index > 0))
    {
      if(hipStreamQuery(mystuff->stream[stream]) == hipSuccess)
      {
#ifdef DEBUG_STREAM_SCHEDULE
        printf(" STREAM_SCHEDULE: found empty stream: = %d (this releases h_ktab[%d])\n", stream, h_ktab_inuse[stream]);
#endif
        h_ktab_index--;
        i                        = h_ktab_inuse[stream];
        h_ktab_inuse[stream]     = h_ktab_cpu[h_ktab_index];
        h_ktab_cpu[h_ktab_index] = i;

        hipMemcpyAsync(mystuff->d_ktab[stream], mystuff->h_ktab[h_ktab_inuse[stream]], size, hipMemcpyHostToDevice, mystuff->stream[stream]);

        k_base.d0 =  k_min_grid[h_ktab_index] & 0xFFFFFFFF;
        k_base.d1 =  k_min_grid[h_ktab_index] >> 32;
        k_base.d2 = 0;

        MFAKTC_FUNC<<<blocksPerGrid, threadsPerBlock, 0, mystuff->stream[stream]>>>(mystuff->exponent, abs_base, is_base_negative, k_base, mystuff->d_ktab[stream], shiftcount, b_preinit, mystuff->d_RES
#ifdef DEBUG_GPU_MATH
                                                                                    , mystuff->d_modbasecase_debug
#endif
                                                                                    );

#ifdef DEBUG_STREAM_SCHEDULE
        printf(" STREAM_SCHEDULE: started GPU kernel on stream %d using h_ktab[%d]\n\n", stream, h_ktab_inuse[stream]);
#endif
#ifdef DEBUG_GPU_MATH
        hipDeviceSynchronize(); /* needed to get the output from device printf() */
#endif
#ifdef DEBUG_STREAM_SCHEDULE_CHECK
        int j, index_count;
        for(i=0; i < (mystuff->num_streams + mystuff->cpu_streams); i++)
        {
          index_count = 0;
          for(j=0; j<mystuff->num_streams; j++)if(h_ktab_inuse[j] == i)index_count++;
          for(j=0; j<mystuff->cpu_streams; j++)if(h_ktab_cpu[j] == i)index_count++;
          if(index_count != 1)
          {
            printf("DEBUG_STREAM_SCHEDULE_CHECK: ERROR: index %d appeared %d times\n", i, index_count);
            printf("  h_ktab_inuse[] =");
            for(j=0; j<mystuff->num_streams; j++)printf(" %d", h_ktab_inuse[j]);
            printf("\n  h_ktab_cpu[] =");
            for(j=0; j<mystuff->cpu_streams; j++)printf(" %d", h_ktab_cpu[j]);
            printf("\n");
          }
        }
#endif
      }
      stream++;
    }
  }

/* wait to finish the current calculations on the device */
  cuda_ret = hipDeviceSynchronize();
  if(cuda_ret != hipSuccess)printf("per class final hipDeviceSynchronize failed!\n");

/* download results from GPU */
  hipMemcpy(mystuff->h_RES, mystuff->d_RES, 32*sizeof(unsigned int), hipMemcpyDeviceToHost);

#ifdef DEBUG_GPU_MATH
  hipMemcpy(mystuff->h_modbasecase_debug, mystuff->d_modbasecase_debug, 32*sizeof(int), hipMemcpyDeviceToHost);
  for(i=0;i<32;i++)if(mystuff->h_modbasecase_debug[i] != 0)printf("h_modbasecase_debug[%2d] = %u\n", i, mystuff->h_modbasecase_debug[i]);
#endif

  mystuff->stats.grid_count = count;
  mystuff->stats.class_time = timer_diff(&timer)/1000;
/* prevent division by zero if timer resolution is too low */
  if(mystuff->stats.class_time == 0)mystuff->stats.class_time = 1;


  if(count > 2 * mystuff->num_streams)mystuff->stats.cpu_wait = (float)twait / ((float)mystuff->stats.class_time * 10);
  else                                mystuff->stats.cpu_wait = -1.0f;

  print_status_line(mystuff);

  if(mystuff->stats.cpu_wait >= 0.0f)
  {
/* if SievePrimesAdjust is enable lets try to get 2 % < CPU wait < 6% */
    if(mystuff->sieve_primes_adjust == 1 && mystuff->stats.cpu_wait > 6.0f && mystuff->sieve_primes < mystuff->sieve_primes_upper_limit && (mystuff->mode != MODE_SELFTEST_SHORT))
    {
      mystuff->sieve_primes *= 9;
      mystuff->sieve_primes /= 8;
      if(mystuff->sieve_primes > mystuff->sieve_primes_upper_limit) mystuff->sieve_primes = mystuff->sieve_primes_upper_limit;
    }
    if(mystuff->sieve_primes_adjust == 1 && mystuff->stats.cpu_wait < 2.0f  && mystuff->sieve_primes > mystuff->sieve_primes_min && (mystuff->mode != MODE_SELFTEST_SHORT))
    {
      mystuff->sieve_primes *= 7;
      mystuff->sieve_primes /= 8;
      if(mystuff->sieve_primes < mystuff->sieve_primes_min) mystuff->sieve_primes = mystuff->sieve_primes_min;
    }
  }


  factorsfound=mystuff->h_RES[0];
  for(i=0; (i<factorsfound) && (i<10); i++)
  {
    factor.d2=mystuff->h_RES[i*3 + 1];
    factor.d1=mystuff->h_RES[i*3 + 2];
    factor.d0=mystuff->h_RES[i*3 + 3];
    print_dez96(factor,string);
    print_factor(mystuff, i, string);
  }
  if(factorsfound>=10)
  {
    print_factor(mystuff, factorsfound, NULL);
  }

  return factorsfound;
}

#undef MFAKTC_FUNC

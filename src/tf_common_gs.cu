#include "hip/hip_runtime.h"
/*
This file is part of mfaktc.
Copyright (C) 2009, 2010, 2011, 2012, 2014, 2015  Oliver Weihe (o.weihe@t-online.de)

mfaktc is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

mfaktc is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with mfaktc.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "gpusieve.h"

#undef RAW_GPU_BENCH // FIXME


#ifdef SHORTCUT_64BIT
extern "C" __host__ int tf_class_64_gs(unsigned long long int k_min, unsigned long long int k_max, mystuff_t *mystuff)
#define MFAKTC_FUNC mfaktc_64_gs
#elif defined (SHORTCUT_75BIT)
extern "C" __host__ int tf_class_75_gs(unsigned long long int k_min, unsigned long long int k_max, mystuff_t *mystuff)
#define MFAKTC_FUNC mfaktc_75_gs
#else
extern "C" __host__ int tf_class_95_gs(unsigned long long int k_min, unsigned long long int k_max, mystuff_t *mystuff)
#define MFAKTC_FUNC mfaktc_95_gs
#endif
{
  int i;
  timeval timer;
  int96 factor,k_base;
  int192 b_preinit;
  int base = mystuff->base;
  int shiftcount, logb, maxlogb, count = 0; // logarithm to base ´base´
  int numblocks;
  unsigned long long k_remaining;
  char string[50];
  int shared_mem_required;
  int factorsfound = 0;

  // If we've never initialized the GPU sieving code, do so now.
//  gpusieve_init (mystuff); // moved to main() function!

  // If we haven't initialized the GPU sieving code for this Mersenne exponent, do so now.
  gpusieve_init_exponent (mystuff);

  // Init the timer
  timer_init(&timer);

  // Pre-calculate some values

  shiftcount=0;
  // shiftcount = (int)log2(exponent), how many bits are there to process
  while((1ULL<<shiftcount) < (unsigned long long int)mystuff->exponent)shiftcount++;
//  printf("\n\nshiftcount = %d\n",shiftcount);
  shiftcount-=1;logb=1;
  maxlogb = shiftcount-3;
  // TODO: find maximum working numbers
#if defined (SHORTCUT_75BIT) || defined (SHORTCUT_64BIT)
  if (maxlogb>16) maxlogb=16; // maximum preprocessing which is possible for 64 bit
#else
  if (maxlogb>20) maxlogb=20; // maximum preprocessing which is possible
#endif
  while(logb<maxlogb || 10*logb<mystuff->bit_min*3)	// how much preprocessing is possible
  {
    shiftcount--;
    logb<<=1; // log(x^2)
    if(mystuff->exponent&(1<<(shiftcount)))logb++; // optional mul with base
  }
//  printf("shiftcount = %d\n",shiftcount);
//  printf("logb = %d\n",logb);
b_preinit.d5=0;b_preinit.d4=0;b_preinit.d3=0;b_preinit.d2=0;b_preinit.d1=0;b_preinit.d0=1;
// just calculate base^logb
#ifdef SHORTCUT_64BIT
  for(i=0; i<logb; i++) mul64(&b_preinit, b_preinit, base);
#elif defined (SHORTCUT_75BIT)
  for(i=0; i<logb; i++) mul75(&b_preinit, b_preinit, base);
#else
  for(i=0; i<logb; i++) mul96(&b_preinit, b_preinit, base);
#endif

/* set result array to 0 */
  hipMemset(mystuff->d_RES, 0, 1*sizeof(int)); //first int of result array contains the number of factors found

#ifdef DEBUG_GPU_MATH
  hipMemset(mystuff->d_modbasecase_debug, 0, 32*sizeof(int));
#endif

  // Calculate the initial bit-to-clear values for this class
  gpusieve_init_class (mystuff, k_min);

  // Generously estimate the shared memory requirements for the TF kernel
#ifdef RAW_GPU_BENCH
  shared_mem_required = 100;						// no sieving = 100%
#else
  if (mystuff->gpu_sieve_primes < 54) shared_mem_required = 100;	// no sieving = 100%
  else if (mystuff->gpu_sieve_primes < 310) shared_mem_required = 50;	// 54 primes expect 48.30%
  else if (mystuff->gpu_sieve_primes < 1846) shared_mem_required = 38;	// 310 primes expect 35.50%
  else if (mystuff->gpu_sieve_primes < 21814) shared_mem_required = 30;	// 1846 primes expect 28.10%
  else if (mystuff->gpu_sieve_primes < 67894) shared_mem_required = 24;	// 21814 primes expect 21.93%
  else shared_mem_required = 22;					// 67894 primes expect 19.94%
#endif
  shared_mem_required = mystuff->gpu_sieve_processing_size * sizeof (int) * shared_mem_required / 100;

  // FIXME: can't use all the shared memory for GPU sieve, lets keep 1kiB spare...
  if(mystuff->verbosity >= 3)printf("shared_mem_required = %d bytes\n", shared_mem_required + 1024);

  if((shared_mem_required + 1024) > mystuff->max_shared_memory)
  {
    printf("ERROR: Not enough shared memory available!\n");
    printf("       Need %d bytes\n", shared_mem_required + 1024);
    printf("       You can lower GPUSieveProcessSize or increase GPUSievePrimes to lower\n");
    printf("       the amount of shared memory needed\n");
    exit(1);
  }


  // Loop until all the k's are processed
  for(;;)
  {

    // Calculate the number of k's remaining.  Round this up so that we sieve an array that is
    // a multiple of the bits processed by each TF kernel (my_stuff->gpu_sieve_processing_size).

    k_remaining = ((k_max - k_min + 1) + NUM_CLASSES - 1) / NUM_CLASSES;
    if (k_remaining < (unsigned long long) mystuff->gpu_sieve_size) {
      numblocks = ((int) k_remaining + mystuff->gpu_sieve_processing_size - 1) / mystuff->gpu_sieve_processing_size;
      k_remaining = numblocks * mystuff->gpu_sieve_processing_size;
    } else
      numblocks = mystuff->gpu_sieve_size / mystuff->gpu_sieve_processing_size;

    // Do some sieving on the GPU.

    gpusieve (mystuff, k_remaining);

    // Set the k value corresponding to the first bit in the bit array

    k_base.d0 = (int) (k_min & 0xFFFFFFFF);
    k_base.d1 = (int) (k_min >> 32);
    k_base.d2 = 0;

    // Now let the GPU trial factor the candidates that survived the sieving

    MFAKTC_FUNC<<<numblocks, THREADS_PER_BLOCK, shared_mem_required>>>(
      mystuff->exponent, mystuff->base, k_base, mystuff->d_bitarray,
      mystuff->gpu_sieve_processing_size, shiftcount,
      b_preinit, mystuff->d_RES
#ifdef DEBUG_GPU_MATH
      , mystuff->d_modbasecase_debug
#endif
      );

    // Sync before doing more GPU sieving
    hipDeviceSynchronize();

    // Count the number of blocks processed
    count += numblocks;

    // Move to next batch of k's
    k_min += (unsigned long long) mystuff->gpu_sieve_size * NUM_CLASSES;
    if (k_min > k_max) break;

    //BUG - we should call a different routine to advance the bit-to-clear values by gpusieve_size bits
    // This will be cheaper than recomputing the bit-to-clears from scratch
    // HOWEVER, the self-test code will ot check this new code unless we make the gpusieve_size much smaller
    gpusieve_init_class (mystuff, k_min);
  }

/* download results from GPU */
  hipMemcpy(mystuff->h_RES, mystuff->d_RES, 32*sizeof(int), hipMemcpyDeviceToHost);

#ifdef DEBUG_GPU_MATH
  hipMemcpy(mystuff->h_modbasecase_debug, mystuff->d_modbasecase_debug, 32*sizeof(int), hipMemcpyDeviceToHost);
  for(i=0;i<32;i++)if(mystuff->h_modbasecase_debug[i] != 0)printf("h_modbasecase_debug[%2d] = %u\n", i, mystuff->h_modbasecase_debug[i]);
#endif

  // Set grid count to the number of blocks processed.  The print code will convert this to a
  // count of candidates processed (by multiplying by 8192 * THREADS_PER_BLOCK.
  // This count isn't an exact match to CPU sieving case as that counts candidates after sieving
  // and we are counting candidates before sieving.  We'd have to modify the TF kernels to count
  // the candidates processed to be completely compatible.
  mystuff->stats.grid_count = count;

  // Keep track of time spent TFing this class
  /* prevent division by zero if timer resolution is too low */
  mystuff->stats.class_time = timer_diff(&timer)/1000;
  if(mystuff->stats.class_time == 0)mystuff->stats.class_time = 1;

  // GPU sieving does not wait on the CPU (also used by print_status_line to indicate this is a GPU sieving kernel)
  mystuff->stats.cpu_wait = -2.0f;

  // Print out a useful status line
  print_status_line(mystuff);

  // Print out any found factors
  factorsfound=mystuff->h_RES[0];
  for(i=0; (i<factorsfound) && (i<10); i++)
  {
    factor.d2=mystuff->h_RES[i*3 + 1];
    factor.d1=mystuff->h_RES[i*3 + 2];
    factor.d0=mystuff->h_RES[i*3 + 3];
    print_dez96(factor,string);
    print_factor(mystuff, i, string);
  }
  if(factorsfound>=10)
  {
    print_factor(mystuff, factorsfound, NULL);
  }

  return factorsfound;
}

#undef MFAKTC_FUNC

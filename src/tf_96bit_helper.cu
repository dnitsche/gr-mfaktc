#include "hip/hip_runtime.h"
/*
This file is part of mfaktc.
Copyright (C) 2009, 2010, 2011, 2012, 2013  Oliver Weihe (o.weihe@t-online.de)

mfaktc is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

mfaktc is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.barrett

You should have received a copy of the GNU General Public License
along with mfaktc.  If not, see <http://www.gnu.org/licenses/>.
*/


__device__ static void check_factor96(int96 f, int96 a, bool negativeBase, unsigned int *RES)
/* Check whether f is a factor or not. If f != 1 and a == 1 then f is a factor,
in this case f is written into the RES array. */
{
  unsigned int index;
  bool isFactor;
  if (negativeBase)
  {
    isFactor = a.d2 == f.d2 && a.d1 == f.d1 && a.d0 == (f.d0 - 1);
  } else
  {
    isFactor = (a.d2|a.d1) == 0 && a.d0 == 1;
  }
  if(isFactor)
  {
    if(f.d2 != 0 || f.d1 != 0 || f.d0 != 1)	/* 1 isn't really a factor ;) */
    {
      index=atomicInc(&RES[0], 10000);
      if(index < 10)				/* limit to 10 factors per class */
      {
        RES[index * 3 + 1] = f.d2;
        RES[index * 3 + 2] = f.d1;
        RES[index * 3 + 3] = f.d0;
      }
    }
  }
}


__device__ static void create_FC96(int96 *f, unsigned int exp, int96 k, unsigned int k_offset)
/* calculates f = 2 * (k+k_offset) * exp + 1 */
{
  int96 exp96;

  exp96.d1 = exp >> 31;
  exp96.d0 = exp << 1;			// exp96 = 2 * exp

  k.d0 = __add_cc (k.d0, __umul32  (k_offset, NUM_CLASSES));
  k.d1 = __addc   (k.d1, __umul32hi(k_offset, NUM_CLASSES));	/* k is limited to 2^64 -1 so there is no need for k.d2 */

  f->d0 = 1 +                                  __umul32(k.d0, exp96.d0);
  f->d1 = __add_cc(__umul32hi(k.d0, exp96.d0), __umul32(k.d1, exp96.d0));
  f->d2 = __addc  (__umul32hi(k.d1, exp96.d0),                        0);

  if(exp96.d1) /* exp96.d1 is 0 or 1 */
  {
    f->d1 = __add_cc(f->d1, k.d0);
    f->d2 = __addc  (f->d2, k.d1);
  }							// f = 2 * k * exp + 1
}


__device__ static void create_FC96_mad(int96 *f, unsigned int exp, int96 k, unsigned int k_offset)
/* similar to create_FC96(), this versions uses multiply-add with carry which
is faster for _SOME_ kernels. */
{
#if (__CUDA_ARCH__ < FERMI) || (CUDART_VERSION < 4010) /* multiply-add with carry is not available on CC 1.x devices and before CUDA 4.1 */
  create_FC96(f, exp, k, k_offset);
#else
  int96 exp96;

  exp96.d1 = exp >> 31;
  exp96.d0 = exp << 1;			// exp96 = 2 * exp

  k.d0 = __umad32_cc(k_offset, NUM_CLASSES, k.d0);
  k.d1 = __umad32hic(k_offset, NUM_CLASSES, k.d1);

  /* umad32 is slower here?! */
  f->d0 = 1 +                                  __umul32(k.d0, exp96.d0);
  f->d1 = __add_cc(__umul32hi(k.d0, exp96.d0), __umul32(k.d1, exp96.d0));
  f->d2 = __addc  (__umul32hi(k.d1, exp96.d0),                        0);

  if(exp96.d1) /* exp96.d1 is 0 or 1 */
  {
    f->d1 = __add_cc(f->d1, k.d0);
    f->d2 = __addc  (f->d2, k.d1);
  }							// f = 2 * k * exp + 1
#endif
}

